
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <sys/time.h>

#define NLINHAS 6144
#define NCOLUNAS 6144
#define THREADS_PER_BLOCK 1024
//32x32
#define NTHREADS 32

__global__ void vector_mul(int *a, int *b, int *c) {
    int i, soma = 0;

    int coluna = blockIdx.x * blockDim.x + threadIdx.x;
    int linha =  blockIdx.y * blockDim.y + threadIdx.y;

    int beginA = NLINHAS * linha;
    int beginB = coluna;

    for (i = 0; i < NLINHAS; i++) {
        soma += a[beginA + i] * b[NLINHAS * i + beginB];
    }

    //printf("%d %d\n", linha, coluna);
    c[linha * NLINHAS + coluna] = soma;
}

int main(){
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;
    int size = NLINHAS * NCOLUNAS * sizeof(int);
    int i, j, n;

    struct timeval timevalA;
	struct timeval timevalB;

    hipMalloc((void **) &d_a, size);
    hipMalloc((void **) &d_b, size);
    hipMalloc((void **) &d_c, size);

    a = (int *)malloc(size);
    b = (int *)malloc(size);
    c = (int *)malloc(size);

    for(i = 0; i < NLINHAS*NCOLUNAS; i++){
//        a[i] = b[i] = i % 10;
        c[i] = 0;
    }

    scanf("%d", &n);

    for (int i = 0; i < NLINHAS; i++) {
        for (j = 0; j < NLINHAS; j++) {
            scanf("%d", &a[i * NLINHAS + j]);
        }
    }

    for (int i = 0; i < NLINHAS; i++) {
        for (j = 0; j < NLINHAS; j++) {
            scanf("%d", &b[i * NLINHAS + j]);
        }
    }

    gettimeofday(&timevalA,NULL);
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    dim3 tbloco = dim3(NLINHAS/NTHREADS, NLINHAS/NTHREADS,1);
    dim3 tthreads = dim3(NTHREADS, NTHREADS, 1);
    vector_mul<<<tbloco,tthreads>>>(d_a, d_b, d_c);
    
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
    gettimeofday(&timevalB,NULL);
    
    // imprimir primeira coluna
    // for (i = 0; i < NLINHAS; i++) {
    //     for (j = 0; j < NLINHAS; j++) {
    //         printf("%d ", c[i * NLINHAS + j]);
    //     }
    //     printf("\n");
    // }
    //printf("\n");

    printf("%.5lf\n", timevalB.tv_sec-timevalA.tv_sec+(timevalB.tv_usec-timevalA.tv_usec)/(double)1000000);

    free(a); free(b); free(c);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
